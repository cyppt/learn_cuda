#include "hip/hip_runtime.h"
#include<add1.h>
#include<iostream>
#include<hip/hip_runtime.h>
#include<>

__global__ void add(int *a, int *b, int *c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while(tid < N)
    {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

extern "C" bool addKernel(int *a, int *b, int *c)
{
    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    add<<<128,128>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return true;
}
