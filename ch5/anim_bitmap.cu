#include "hip/hip_runtime.h"
#include<anim_bitmap.h>

#include<cstdlib>
#include<cmath>
#include<iostream>
#include<hip/hip_runtime.h>
#include<>

__global__ void kernel(unsigned char *ptr, int ticks)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x; //计算x为第几个线程
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x; // 计算 x,y 的索引

    float fx = x - DIM/2;
    float fy = y - DIM/2;
    float d = sqrtf(fx * fx + fy * fy);
    unsigned char grey = (unsigned char)(128.0f + 127.0f * cos(d/10.0f - ticks/7.0f)/(d/10.0f + 1.0f));
    ptr[offset + 0] = grey;
    ptr[offset + 1] = grey;
    ptr[offset + 2] = grey;
    ptr[offset + 3] = 255;
}

void cleanup(DataBlock *d)
{
    hipFree(d->dev_bitmap);
}

void generate_frame(DataBlock *d, int ticks)
{
    dim3 blocks(DIM/16,DIM/16);
    dim3 threads(16,16);
    kernel<<<blocks, threads>>>(d->dev_bitmap, ticks);
    hipMemcpy(d->bitmap->get_ptr(),d->dev_bitmap,d->bitmap->image_size(),hipMemcpyDeviceToHost);
}

extern "C" bool bitmapKernel(DataBlock *data)
{
    CPUAnimBitmap bitmap(DIM,DIM,(void*)data);
    data->bitmap = &bitmap;
    hipMalloc((void**)&data->dev_bitmap, bitmap.image_size());
    bitmap.anim_and_exit((void(*)(void*,int))generate_frame, (void(*)(void*))cleanup);
    
    return true;
}