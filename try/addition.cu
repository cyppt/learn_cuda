#include "hip/hip_runtime.h"
#include <addition.h>
 __global__ void add(int *a, int *b, int *c)   //在设备用运行的
 {
     *c=*a+*b;
 }
 
 extern "C" bool addition(int a, int b, int *c)
 {
     int *d_a, *d_b, *d_c;
     int size=sizeof(int);
     
     hipMalloc((void **)&d_a, size);
     hipMalloc((void **)&d_b, size);
     hipMalloc((void **)&d_c, size);
     
     hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);  //不能在主机代码中对hipMalloc，分配的指针进行内存读写操作
     hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice); // 源指针位于主机上，传递给设备，可以反过来，也可DeviceToDevice
     
     add<<<1,1>>>(d_a, d_b, d_c);  //设备函数调用

    //  cudaDevicePror pror;
    //  int count;
    //  hipGetDeviceCount(&count);
    //  for (int i = 0; i < count;i++)
    //  {
    //     hipGetDeviceProperties(&prop, i);
    //     printf(" --- General Information for device %d --- \n", i);
    //     printf("Name: %s\n", prop.name);
    //     printf("Compute capability: %d.%      d\n", prop.major, prop.minor);
        
    //  }
     
     hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
     
     hipFree(d_a); hipFree(d_b); hipFree(d_c);
     return true;
}