#include "hip/hip_runtime.h"
#include<ray_trace_2.h>

__constant__ Sphere s[SPHERES]; // 常量内存

__global__ void kernel(unsigned char *ptr)
{   
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    // 图像坐标偏移，使z过原点
    float ox = (x - DIM/2);
    float oy = (y - DIM/2);
    float r = 0, g = 0, b = 0;
    float maxz = -INF; // 具体原因见readme
    for (int i = 0; i < SPHERES; i++)
    {
        float n;
        float t = s[i].hit(ox, oy, &n);
        if (t > maxz)
        {
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t;
        }
    }
    ptr[offset * 4 + 0] = int(r * 255);
    ptr[offset * 4 + 1] = int(g * 255);
    ptr[offset * 4 + 2] = int(b * 255);
    ptr[offset * 4 + 3] = 255;
}

extern "C" void ray_trace()
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    CPUBitmap bitmap(DIM, DIM);
    unsigned char *dev_bitmap;
    Sphere *temp_s;

    temp_s = (Sphere*)malloc(sizeof(Sphere) * SPHERES);
    for(int i = 0; i < SPHERES; i++)
    {
        temp_s[i].r = rnd(1.0f);
        temp_s[i].g = rnd(1.0f);
        temp_s[i].b = rnd(1.0f);
        temp_s[i].x = rnd(1000.0f) - 500;
        temp_s[i].y = rnd(1000.0f) - 500;
        temp_s[i].z = rnd(1000.0f) - 500;
        temp_s[i].radius = rnd(100.0f) + 20;
    }

    hipMalloc((void**)&dev_bitmap, bitmap.image_size());

    hipMemcpyToSymbol(HIP_SYMBOL(s),temp_s, sizeof(Sphere) * SPHERES); // 常量内存 直接hipMemcpy 不用malloc和free

    dim3 grids(DIM/16, DIM/16);
    dim3 threads(16,16);

    hipEventRecord(start , 0);
    kernel<<<grids, threads>>>(dev_bitmap);
    hipEventRecord(stop , 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time to generate:" << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);
    bitmap.display_and_exit();

    hipFree(dev_bitmap);
    free(temp_s);
}